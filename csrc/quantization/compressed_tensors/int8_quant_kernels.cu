#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <torch/all.h>
#include <cmath>

#include "../../dispatch_utils.h"
#include "../../reduction_utils.cuh"

static inline __device__ int8_t float_to_int8_rn(float x) {
#ifdef USE_ROCM
  static const float i8_min =
      static_cast<float>(std::numeric_limits<int8_t>::min());
  static const float i8_max =
      static_cast<float>(std::numeric_limits<int8_t>::max());
  // round
  float dst = std::nearbyint(x);
  // saturate
  dst = std::clamp(dst, i8_min, i8_max);
  return static_cast<int8_t>(dst);
#else
  // CUDA path
  uint32_t dst;
  asm volatile("cvt.rni.sat.s8.f32 %0, %1;" : "=r"(dst) : "f"(x));
  return reinterpret_cast<const int8_t&>(dst);
#endif
}

namespace vllm {

template <typename scalar_t, typename scale_type>
__global__ void static_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type const* scale_ptr, const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;
  scale_type const scale = *scale_ptr;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] = float_to_int8_rn(
        static_cast<float>(input[token_idx * hidden_size + i]) / scale);
  }
}

template <typename scalar_t, typename scale_type>
__global__ void dynamic_scaled_int8_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, const int hidden_size) {
  int const tid = threadIdx.x;
  int const token_idx = blockIdx.x;
  float absmax_val = 0.0f;
  float const zero = 0.0f;

  for (int i = tid; i < hidden_size; i += blockDim.x) {
    float val = static_cast<float>(input[token_idx * hidden_size + i]);
    val = val > zero ? val : -val;
    absmax_val = val > absmax_val ? val : absmax_val;
  }

  float const block_absmax_val_maybe = blockReduceMax(absmax_val);
  __shared__ float block_absmax_val;
  if (tid == 0) {
    block_absmax_val = block_absmax_val_maybe;
    scale[token_idx] = block_absmax_val / 127.0f;
  }
  __syncthreads();

  float const tmp_scale = 127.0f / block_absmax_val;
  for (int i = tid; i < hidden_size; i += blockDim.x) {
    out[token_idx * hidden_size + i] = float_to_int8_rn(
        static_cast<float>(input[token_idx * hidden_size + i]) * tmp_scale);
  }
}

template <typename scalar_t, typename scale_type, typename azp_type>
__global__ void dynamic_scaled_int8_azp_quant_kernel(
    scalar_t const* __restrict__ input, int8_t* __restrict__ out,
    scale_type* scale, azp_type* azp, const int hidden_size) {
  int const token_idx = blockIdx.x;

  // Scan for the min and max value for this token
  float max_val = 0.0f;
  float min_val = std::numeric_limits<float>::max();
  for (int i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    auto val = static_cast<float>(input[token_idx * hidden_size + i]);
    max_val = fmaxf(max_val, val);
    min_val = fminf(min_val, val);
  }

  // Reduce the max and min values across the block
  max_val = blockReduceMax(max_val);
  //  if (threadIdx.x == 0 and blockIdx.x == DEBUG_TOKEN) printf("MIN:");
  min_val = blockReduceMin(min_val);

  __shared__ scale_type scale_sh;
  __shared__ azp_type azp_sh;

  // Compute the scale and zero point and store them, only on the first thread
  if (threadIdx.x == 0) {
    float scale_val = (max_val - min_val) / 255.0f;
    auto const azp_float = roundf(min_val / scale_val + 128.0f);
    auto const azp_val = static_cast<azp_type>(azp_float);

    // Azp was rounded, which may cause the range to be slightly off.
    // Expand the range to make sure all values are representable.
    auto const min_nozp = static_cast<float>(azp_val - 128);
    auto const max_nozp = static_cast<float>(azp_val + 127);
    auto no_div_0 = [&](float num, float div) {
      return div == 0.0f ? scale_val : num / div;
    };

    scale_val = fmaxf(no_div_0(max_val, max_nozp), no_div_0(min_val, min_nozp));

    // Store the scale and azp
    scale[token_idx] = scale_sh = scale_val;
    azp[token_idx] = azp_sh = azp_val;
  }

  // Wait for the scale and azp to be computed
  __syncthreads();

  float const scale_val = scale_sh;
  azp_type const azp_val = azp_sh;

  // Quantize the values
  for (int i = threadIdx.x; i < hidden_size; i += blockDim.x) {
    auto val = static_cast<float>(input[token_idx * hidden_size + i]);
    auto quant_val = static_cast<int8_t>(roundf(val / scale_val) - azp_val);
    out[token_idx * hidden_size + i] = quant_val;
  }
}

}  // namespace vllm

void static_scaled_int8_quant(torch::Tensor& out,          // [..., hidden_size]
                              torch::Tensor const& input,  // [..., hidden_size]
                              torch::Tensor const& scale) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());
  TORCH_CHECK(scale.numel() == 1);

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "static_scaled_int8_quant_kernel", [&] {
        vllm::static_scaled_int8_quant_kernel<scalar_t, float>
            <<<grid, block, 0, stream>>>(input.data_ptr<scalar_t>(),
                                         out.data_ptr<int8_t>(),
                                         scale.data_ptr<float>(), hidden_size);
      });
}

void dynamic_scaled_int8_quant(
    torch::Tensor& out,          // [..., hidden_size]
    torch::Tensor const& input,  // [..., hidden_size]
    torch::Tensor& scales, c10::optional<torch::Tensor> const& azp) {
  TORCH_CHECK(input.is_contiguous());
  TORCH_CHECK(out.is_contiguous());

  int const hidden_size = input.size(-1);
  int const num_tokens = input.numel() / hidden_size;
  dim3 const grid(num_tokens);
  dim3 const block(std::min(hidden_size, 1024));
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  VLLM_DISPATCH_FLOATING_TYPES(
      input.scalar_type(), "dynamic_scaled_int8_quant_kernel", [&] {
        if (!azp) {
          vllm::dynamic_scaled_int8_quant_kernel<scalar_t, float>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), hidden_size);
        } else {
          vllm::dynamic_scaled_int8_azp_quant_kernel<scalar_t, float, int32_t>
              <<<grid, block, 0, stream>>>(
                  input.data_ptr<scalar_t>(), out.data_ptr<int8_t>(),
                  scales.data_ptr<float>(), azp->data_ptr<int32_t>(),
                  hidden_size);
        }
      });
}
